#include "hip/hip_runtime.h"
#include <hopper_gemm.hpp>
#include <hopper_gemm_utils.hpp>

__global__ void hopper_gemm_fp32_kernel(int m, int n, int k, float alpha,
                                        const float* A, int lda, const float* B,
                                        int ldb, float beta, float* C, int ldc)
{
}

void hopper_gemm_fp32(int m, int n, int k, float alpha, const float* A, int lda,
                      const float* B, int ldb, float beta, float* C, int ldc)
{
    dim3 blockDim(32U * 32U, 1U);
    dim3 gridDim(128U, 128U, 1U);
    hopper_gemm_fp32_kernel<<<gridDim, blockDim>>>(m, n, k, alpha, A, lda, B,
                                                   ldb, beta, C, ldc);
}